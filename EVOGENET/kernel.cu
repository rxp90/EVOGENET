#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "hipblas.h"
#include "thrust/sort.h"
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/sequence.h>
#include <thrust/transform_reduce.h>
#include <thrust/extrema.h>
#include <thrust\device_vector.h>

static void HandleError(hipError_t err,
	const char *file,
	int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

template <typename T>
struct inverse
{
	__host__ __device__
		T operator()(const T& x) const {
		return 1 / (pow(x, 1));
	}
};

#ifndef __HIPCC__  
#define __HIPCC__
#endif

#define IDX2C(i, j, ld) (((j)*(ld))+(i))
#define MAX_INPUTS 32

#define POPULATIONS 1
#define RULES_PER_NODE 1
#define NODES 32
#define POPULATION (1024*POPULATIONS)
//#define LAMBDA 0.9
#define ELITE_MEMBERS (POPULATION)
#define ELEMENTS_TO_MIGRATE (2*POPULATIONS)
#define MIGRATION_FREQUENCY 2

#define INDIVIDUALS_PER_BLOCK 2

#define MAX_CONNECTIVITY_DISTANCE 0.1

#define GENERATIONS 200
#define EXECUTIONS 1
#define LINK_MUTATION_PROB 0.001  
#define RULE_MUTATION_PROB 0.001

#define INDIVIDUAL_SIZE (NODES * MAX_INPUTS)

#define ROULETTE 0
#define ELITE 1

#define MOST 0
#define ABSOLUTE_REPRESSOR 1
#define JOINT_ACTIVATORS 2
#define JOINT_REPRESSORS 3

typedef struct
{
	char nodes[NODES];
	char links[NODES*MAX_INPUTS];
	char rules[RULES_PER_NODE*NODES];
} network;

__device__ const network INIT_NETWORK = {
	{ 1, 0, 0, 1, 0, 1, 0, 0, 1, 1, 0, 0, 1, 1, 1, 1, 1, 0, 1, 0, 1, 1, 1, 0, 0, 1, 1, 1, 1, 1, 1, 1 },
	{ 1, 0, 0, 1, 0, 0, 0, -1, -1, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, -1, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 1, 0, 1, 0, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 1, 0, 0, 0, 0, 1, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, -1, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, -1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, -1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, -1, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, -1, 0, 0, -1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 1, 1, 1, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, -1, 0, 1, -1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, -1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 1, 1, 1, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 0, 1, 1, -1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, -1, 0, 0, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, -1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, -1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0 }
};
network INIT_NETWORK_HOST = {
	{ 1, 0, 0, 1, 0, 1, 0, 0, 1, 1, 0, 0, 1, 1, 1, 1, 1, 0, 1, 0, 1, 1, 1, 0, 0, 1, 1, 1, 1, 1, 1, 1 },
	{ 1, 0, 0, 1, 0, 0, 0, -1, -1, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, -1, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 1, 0, 1, 0, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 1, 0, 0, 0, 0, 1, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, -1, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, -1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, -1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, -1, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, -1, 0, 0, -1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 1, 1, 1, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, -1, 0, 1, -1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, -1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 1, 1, 1, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 0, 1, 1, -1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, -1, 0, 0, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, -1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, -1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0 }
};
network GOAL_NETWORK_HOST = {
	{ 1, 1, 1, 0, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 1, 1, 1, 1, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1 },
	{ 1, 0, 0, 1, 0, 0, 0, -1, -1, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, -1, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 1, 0, 1, 0, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 1, 0, 0, 0, 0, 1, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, -1, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, -1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, -1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, -1, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, -1, 0, 0, -1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 1, 1, 1, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, -1, 0, 1, -1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, -1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 1, 1, 1, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 0, 1, 1, -1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, -1, 0, 0, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, -1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, -1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0 }
};

float LAMBDA_HOST[POPULATIONS] = { .9 };

__constant__ float LAMBDA_VALUES[POPULATIONS];

__constant__ float INIT_CONNECTIVITY;

float BEST_FITNESS_HOST = 1.0;

__device__ unsigned int WangHash(unsigned int a){
	a = (a ^ 61) ^ (a >> 16);
	a = a + (a << 3);
	a = a ^ (a >> 4);
	a = a * 0x27d4eb2d;
	a = a ^ (a >> 15);
	return a;
}

__device__ float generate(hiprandState* globalState)
{
	int ind = blockIdx.x*blockDim.x + threadIdx.x;

	hiprandState localState = globalState[ind];
	float RANDOM = hiprand_uniform(&localState);
	globalState[ind] = localState;
	return RANDOM;
}


__device__ void generate_v2(hiprandState* globalState, float * values, unsigned int count)
{
	int ind = blockIdx.x*blockDim.x + threadIdx.x;

	hiprandState localState = globalState[ind];
	for (int i = 0; i < count; i++){
		values[i] = hiprand_uniform(&localState);
	}

	globalState[ind] = localState;

}

__global__ void setup_kernel(hiprandState * state)
{
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int seed = (unsigned int)clock64();
	hiprand_init(seed, id, 0, &state[id]);
}

__global__ void setup_kernel_V3(hiprandState * state)
{
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int seed = (unsigned int)clock64();
	hiprand_init(WangHash(seed) + id, 0, 0, &state[id]);
}


__device__  void shuffle(char *array, int n, hiprandState *globalState)
{
	int i = n - 1;
	int j, temp;
	while (i > 0)
	{
		j = generate(globalState)*(i + 1);
		temp = array[i];
		array[i] = array[j];
		array[j] = temp;
		i = i - 1;
	}
}


__host__ __device__ void print_array(const char array[], int size)
{
	int i;
	//	printf("{");
	for (i = 0; i < size - 1; i++)
	{
		printf("%d,", array[i]);
	}
	printf("%d", array[size - 1]);
	printf("\n");
}
__host__ __device__ void print_array_f(const float array[], int size)
{
	int i;
	printf("[");
	for (i = 0; i < size; i++)
	{
		printf("%10f", array[i]);
	}
	printf("]\n");
}
__host__ void print_array_file(const char array[], int size, FILE *f)
{
	int i;
	fprintf(f, "{%d", array[0]);
	for (i = 1; i < size; i++)
	{
		fprintf(f, ",%d", array[i]);
	}
	fprintf(f, "};\n");
}


__device__ __host__ void print_network(network individual)
{
	printf("--\n");
	print_array(individual.nodes, NODES);
	print_array(individual.links, NODES * MAX_INPUTS);
	print_array(individual.rules, RULES_PER_NODE * NODES);
}
__host__ void print_network_file(network individual, FILE *f)
{
	fprintf(f, "\nNODES: ");
	print_array_file(individual.nodes, NODES, f);
	fprintf(f, "\nLINKS: ");
	print_array_file(individual.links, NODES * MAX_INPUTS, f);
	fprintf(f, "\nRULES: ");
	print_array_file(individual.rules, RULES_PER_NODE * NODES, f);
}

__device__ __host__ void print_population(network population[], int population_size)
{
	int i;
	printf("------------------- POPULATION -------------------\n");
	for (i = 0; i < population_size; ++i)
	{
		print_network(population[i]);
		//  printf("Fitness value: %f\n",evaluate_individual(&population[i], &GOAL_NETWORK_HOST, MAX_INPUTS, NODES, LAMBDA,0));
	}
	printf("---------------------------------------------------\n");
}
void inline print_progress(float progress, float best)
{
	if (progress < 1.0)
	{
		int barWidth = 50;
		int i;

		printf(" [");
		int pos = barWidth * progress;
		for (i = 0; i < barWidth; ++i)
		{
			if (i < pos) printf("%c", '=');
			else if (i == pos) printf("%c", '>');
			else printf("%c", ' ');
		}
		// printf("] %.1f%\r", progress * 100.0);
		printf("] %.2f -- Best: %.8f\r", progress * 100, best);

		fflush(stdout);
	}
}

void print_parameters(){
	unsigned int width = 35;
	printf("TOTAL POPULATION: %*d\n", width, POPULATION);
	printf("ISLANDS (POPULATIONS): %*d\n", width, POPULATIONS);
	printf("MIGRATIONS/ISLAND: %*d\n", width, ELEMENTS_TO_MIGRATE / POPULATIONS);
	printf("MIGRATION FREQUENCY: %*d\n", width, MIGRATION_FREQUENCY);
	//printf("LAMBDA: \t%*.3f\n", width, LAMBDA);
	printf("LINK MUTATION PROB: %*.3f\n", width, LINK_MUTATION_PROB);
	printf("RULE MUTATION PROB: %*.3f\n", width, RULE_MUTATION_PROB);
	printf("GENERATIONS: %*d\n", width, GENERATIONS);
}


template <unsigned int individuals_per_block>
__device__ void apply_rules(char links[], int links_number, char rule, char nodes[], unsigned int node_index)
{

	__shared__ char input_count[3 * individuals_per_block*NODES];

	char a, count_index;

	char states[3] = { 0, 1, nodes[node_index] };

	char rules_to_states[4];

	for (char j = 0; j < 5; j++){
		for (char i = 0; i < links_number; i++)
		{
			a = (links[i] * nodes[i + (node_index >> 5)*NODES]);

			count_index = (-(a - 1)*(3 * a + 4)) >> 1;

			input_count[count_index + node_index * 3]++;

		}

		char x1 = (input_count[0 + node_index * 3] > input_count[1 + node_index * 3]) - (input_count[0 + node_index * 3] < input_count[1 + node_index * 3]); // (+ > -) - ( + < -)
		rules_to_states[MOST] = (-(x1 + 1)*(3 * x1 - 4)) >> 1;

		char x2 = input_count[1 + node_index * 3] > 0;
		rules_to_states[ABSOLUTE_REPRESSOR] = 2 - (x2 << 1);

		char x3 = input_count[0 + node_index * 3] > 1;
		rules_to_states[JOINT_ACTIVATORS] = 2 - x3;

		char x4 = input_count[1 + node_index * 3] > 1;
		rules_to_states[JOINT_REPRESSORS] = 2 - (x4 << 1);

		nodes[node_index] = states[rules_to_states[rule]];
		__syncthreads(); // Not needed but increases performance. Due to bank conflicts?
	}

}

__global__ void link_distance(network population[], float *distances, int size, char goal_links[]){

	const unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;
	const unsigned int individual = index / INDIVIDUAL_SIZE;
	const unsigned int link = index % INDIVIDUAL_SIZE;
	const unsigned int population_index = individual / (POPULATION / POPULATIONS);

	if (individual < POPULATION){
		float distance = (population[individual].links[link] != goal_links[link]);
		distance *= (1 - LAMBDA_VALUES[population_index]) / (NODES*MAX_INPUTS);
		distances[IDX2C(individual, link, POPULATION)] = distance;
	}
}
template <unsigned int links_per_node>
__global__ void link_distance_v2(network population[], float *distances, int size, char goal_links[]){

	const unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;
	const unsigned int individual = (index*links_per_node) / INDIVIDUAL_SIZE;
	const unsigned int link = (index*links_per_node) % INDIVIDUAL_SIZE;
	const unsigned int population_index = individual / (POPULATION / POPULATIONS);

	char links[links_per_node];
	char goal[links_per_node];

	if (individual < POPULATION){
		int2 *p = reinterpret_cast<int2*>(population[individual].links + link);
		int2 links_vec = p[0];
		reinterpret_cast<int2*>(links)[0] = links_vec;

		int2 *p2 = reinterpret_cast<int2*>(goal_links + link);
		int2 goal_vec = p2[0];
		reinterpret_cast<int2*>(goal)[0] = goal_vec;
		for (char i = 0; i < links_per_node; i++){
			float distance = (links[i] != goal[i]);
			distance *= (1 - LAMBDA_VALUES[population_index]) / (NODES*MAX_INPUTS);
			distances[IDX2C(individual, link + i, POPULATION)] = distance;
		}
	}
}
template <unsigned int individuals_per_block>
__global__ void node_distance(network population[], float *distances, int size, char init_nodes[], char goal_nodes[]){

	const unsigned char node = threadIdx.x % NODES;

	__shared__ char nodes[individuals_per_block*NODES];
	__shared__ unsigned int offset;

	char node_inputs[MAX_INPUTS];

	if (threadIdx.x == 0){
		offset = blockIdx.x * individuals_per_block;
	}

	__syncthreads();

	nodes[threadIdx.x] = init_nodes[node];

	const unsigned int individual = (threadIdx.x / NODES) + offset;
	const unsigned int population_index = individual / (POPULATION / POPULATIONS);

	if (individual < POPULATION){

		const unsigned char rule = population[individual].rules[node];

		//TODO En la primera iteracion puedo calcular directamente represores y activadores
		for (int i = 0; i < 2; i++){
			int4 *p = reinterpret_cast<int4*>(population[individual].links + node*MAX_INPUTS + i * 16);
			int4 links_vec = p[0];
			reinterpret_cast<int4*>(node_inputs)[i] = links_vec;
		}

		apply_rules<individuals_per_block>(node_inputs, MAX_INPUTS, rule, nodes, threadIdx.x);

		float distance = (nodes[threadIdx.x] != goal_nodes[node]);
		distance *= LAMBDA_VALUES[population_index] / NODES;
		distances[IDX2C(individual, node, POPULATION)] = distance;
	}
}

hipblasStatus_t sum_link_distances(const float* A, const float* d_x, float* d_y, const int row, const int col, hipblasHandle_t handle){

	// level 2 calculation y = alpha * A * x + beta * y
	float alf = 1.f;
	float beta = 0.f;

	return hipblasSgemv(handle, HIPBLAS_OP_N, col, row, &alf, A, col, d_x, 1, &beta, d_y, 1);//swap col and row
}

hipblasStatus_t sum_fitness(const float* fitness, float* result, hipblasHandle_t handle){

	return hipblasSasum(handle, POPULATION, fitness, 1, result);
}

hipblasStatus_t sum_node_distances(const float* A, const float* d_x, float* d_y, const int row, const int col, hipblasHandle_t handle){

	// level 2 calculation y = alpha * A * x + beta * y
	float alf = 1.f;
	float beta = 0.f;

	return hipblasSgemv(handle, HIPBLAS_OP_N, col, row, &alf, A, col, d_x, 1, &beta, d_y, 1);//swap col and row

}
hipblasStatus_t pop_fitness(const float* link_distances, const float* node_distances, float* fitness, hipblasHandle_t handle){

	// level 2 calculation y = alpha * A * x + beta * y
	float alf = 1.f;
	float beta = 1.f;
	return hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, POPULATION, 1, &alf, link_distances, POPULATION, &beta, node_distances, POPULATION, fitness, POPULATION);

}

__device__ void generate_rules(char rules[], int size, hiprandState * globalState)
{
	int i, rule;
	float randoms[RULES_PER_NODE*NODES];
	generate_v2(globalState, randoms, RULES_PER_NODE*NODES);
	for (i = 0; i < size; i++)
	{
		rule = randoms[i] * 4;
		rules[i] = rule; /* Four possible rules */
	}
}


__global__ void generate_individual(network population[], int nodes, int max_inputs, int rules_per_node, hiprandState* globalState)
{

	int i = threadIdx.x + blockDim.x*blockIdx.x;
	// TODO �individuo en local?
	if (i < POPULATION){
		population[i] = INIT_NETWORK;
		shuffle(population[i].links, max_inputs * nodes, globalState);
		generate_rules(population[i].rules, rules_per_node * nodes, globalState);
	}

}

__global__ void mutation(network *population, int nodes, int max_inputs_per_node, int rules_per_node, float link_mut_prob, float rule_mut_prob, hiprandState *globalState)
{

	const unsigned int individual_index = (blockIdx.x*blockDim.x + threadIdx.x);

	const char rules[] = { 1, 2, 3, 0, 2, 3, 0, 1, 3, 0, 1, 2 };
	const char links[] = { 0, 1, 1, -1, 0, -1 };

	char link_to_change, rule_to_change;
	int link_index, rule_index;
	int new_link_index, new_rule_index;

	float randoms[6];
	generate_v2(globalState, randoms, 6);

	if (individual_index < POPULATION){

		if (randoms[0] <= link_mut_prob)
		{
			link_index = randoms[1] * (nodes * max_inputs_per_node);
			link_to_change = population[individual_index].links[link_index];
			new_link_index = randoms[2] * 2 + (link_to_change + 1) * 2;
			population[individual_index].links[link_index] = links[new_link_index];
		}

		if (randoms[3] <= rule_mut_prob)
		{
			rule_index = randoms[4] * rules_per_node * nodes;
			rule_to_change = population[individual_index].rules[rule_index];
			new_rule_index = randoms[5] * 3 + rule_to_change * 3;
			population[individual_index].rules[rule_index] = rules[new_rule_index];
		}

	}

}


template <unsigned int blockSize>
__device__ void calculeConnectivity(volatile char links[], int size, float *connectivity){

	const unsigned int tid = threadIdx.x;

	__shared__ char sdata[INDIVIDUAL_SIZE / 2];

	int mySum;

	if (tid < size){
		mySum = links[tid] != 0;
	}

	if (tid + blockSize < size){
		mySum += (int)(links[tid + blockSize] != 0);
	}

	sdata[tid] = mySum;
	__syncthreads();

	// do reduction in shared mem
	if ((blockSize >= 512) && (tid < 256))
	{
		sdata[tid] = mySum = mySum + sdata[tid + 256];
	}

	__syncthreads();

	if ((blockSize >= 256) && (tid < 128))
	{
		sdata[tid] = mySum = mySum + sdata[tid + 128];
	}

	__syncthreads();

	if ((blockSize >= 128) && (tid < 64))
	{
		sdata[tid] = mySum = mySum + sdata[tid + 64];
	}

	__syncthreads();

#if (__CUDA_ARCH__ >= 300 )
	if (tid < 32)
	{
		// Fetch final intermediate sum from 2nd warp
		if (blockSize >= 64){
			mySum += sdata[tid + 32];
		}
		// Reduce final warp using shuffle
		for (int offset = warpSize / 2; offset > 0; offset /= 2)
		{
			mySum += __shfl_down(mySum, offset);
		}
	}
#else
	// fully unroll reduction within a single warp
	if ((blockSize >= 64) && (tid < 32))
	{
		sdata[tid] = mySum = mySum + sdata[tid + 32];
	}

	__syncthreads();

	if ((blockSize >= 32) && (tid < 16))
	{
		sdata[tid] = mySum = mySum + sdata[tid + 16];
	}

	__syncthreads();

	if ((blockSize >= 16) && (tid < 8))
	{
		sdata[tid] = mySum = mySum + sdata[tid + 8];
	}

	__syncthreads();

	if ((blockSize >= 8) && (tid < 4))
	{
		sdata[tid] = mySum = mySum + sdata[tid + 4];
	}

	__syncthreads();

	if ((blockSize >= 4) && (tid < 2))
	{
		sdata[tid] = mySum = mySum + sdata[tid + 2];
	}

	__syncthreads();

	if ((blockSize >= 2) && (tid < 1))
	{
		sdata[tid] = mySum = mySum + sdata[tid + 1];
	}

	__syncthreads();
#endif

	if (tid == 0){
		*connectivity = ((float)mySum) / size;
	}
	__syncthreads();	// �Necesario?
}

__global__ void sus_selection_v1(float population_fitness[], float total_fitness, int parents, int indices[], hiprandState *globalState)
{
	// Calculate distance between the pointers
	float pointer_distance = total_fitness / parents;
	// Pick random number between 0 and p
	float start = generate(globalState) * pointer_distance;
	int index = 0;
	float sum = population_fitness[index];
	int i;
	float pointer;
	for (i = 0; i < parents / POPULATIONS; i++)
	{
		pointer = start + i*pointer_distance;
		if (sum >= pointer)
		{
			indices[i] = index;
		}
		else
		{
			for (++index; index < POPULATION; index++)
			{
				sum += population_fitness[index];
				if (sum >= pointer)
				{
					indices[i] = index;
					break;
				}
			}
		}
	}
}

__global__ void min(network population[], float* elements, int size, int generation, float *BEST_INDIVIDUAL_FITNESS, network * BEST_INDIVIDUAL){

	float min = 1;
	int index = 0;
	for (int i = 0; i<size; i++){
		if (min>elements[i]){
			min = elements[i];
			index = i;
		}
	}
	if (min < *(BEST_INDIVIDUAL_FITNESS)){
		*BEST_INDIVIDUAL_FITNESS = min;
		*BEST_INDIVIDUAL = population[index];
		//	print_network(population[index]);
	}

	//print_network(population[index]);
}

void min_v2(network population[], thrust::device_ptr<float> fitness, thrust::device_ptr<float> min_ptr, int generation, FILE *f, float * best_fitness, network * best_individual){

	float min = min_ptr[0];
	int position = thrust::distance(fitness, min_ptr);

	if (min < *(best_fitness)){
		*best_fitness = min;

		HANDLE_ERROR(
			hipMemcpy(best_individual, population + position, sizeof(network), hipMemcpyDeviceToHost)
			);

	}

	fprintf(f, "%d,%.8f\n", generation, *best_fitness);

}

void sort_population(thrust::device_ptr<float> keys, thrust::device_ptr<int> indices){

	for (int i = 0; i < (POPULATIONS); i++){
		thrust::sort_by_key(keys + i*POPULATION / POPULATIONS, keys + (i + 1)*POPULATION / POPULATIONS, indices + i*POPULATION / POPULATIONS);
	}

}

__global__ void sequence(int * indices){
	int index = threadIdx.x + blockDim.x*blockIdx.x;
	if (index < POPULATION){
		indices[index] = index % (POPULATION / POPULATIONS);
	}
}

__global__ void generateLinkCrossoverindices(int *indices, hiprandState *globalState){
	int index = threadIdx.x + blockDim.x *blockIdx.x;
	int N = ELITE_MEMBERS / 2;
	if (index < N){
		indices[index] = generate(globalState)*NODES*MAX_INPUTS;
	}
}
__global__ void generateRuleCrossoverindices(int *indices, hiprandState *globalState){
	int index = threadIdx.x + blockDim.x *blockIdx.x;
	int N = ELITE_MEMBERS / 2;
	if (index < N){
		indices[index] = generate(globalState)*RULES_PER_NODE*NODES;
	}
}

template <unsigned int blockSize, unsigned int offset>
__global__ void crossover(network population[], int *indices_best, int *link_crossover_indices){

	const unsigned int population_index = (blockIdx.x * 2) / (POPULATION / POPULATIONS);
	/*
	const int index_parent_1 = indices_best[blockIdx.x * 2] + offset*population_index;
	const int index_parent_2 = indices_best[blockIdx.x * 2 + 1] + offset*population_index;
	*/
	int indices[2] = { indices_best[blockIdx.x * 2] + offset*population_index, indices_best[blockIdx.x * 2 + 1] + offset*population_index };

	const unsigned int tid = threadIdx.x;

	// Replace parents!
	/*const int index_replacement_1 = indices[0];
	const int index_replacement_2 = indices[1];
	*/
	__shared__ char links_child_1[INDIVIDUAL_SIZE];
	__shared__ char links_child_2[INDIVIDUAL_SIZE];

	__shared__ unsigned char rules_child_2[RULES_PER_NODE*NODES];

	const int link_crossover_index = link_crossover_indices[blockIdx.x];
	const int rule_crossover_index = (link_crossover_index / NODES) + 1;

	/** Copy the children's links **/

	const char links_condition = (tid < link_crossover_index);

	if (tid < INDIVIDUAL_SIZE){
		links_child_1[tid] = population[indices[!links_condition]].links[tid];
		links_child_2[tid] = population[indices[links_condition]].links[tid];
	}
	if (tid + blockSize < INDIVIDUAL_SIZE){
		links_child_1[tid + blockSize] = population[indices[!links_condition]].links[tid + blockSize];
		links_child_2[tid + blockSize] = population[indices[links_condition]].links[tid + blockSize];
	}

	const char rules_condition = (tid >= rule_crossover_index);

	if (tid < RULES_PER_NODE*NODES){
		rules_child_2[tid] = population[indices[!rules_condition]].rules[tid];
	}
	/*
	if (tid < size){
	if (tid < link_crossover_index){
	links_child_1[tid] = population[index_parent_1].links[tid];
	links_child_2[tid] = population[index_parent_2].links[tid];
	}
	else{
	links_child_1[tid] = population[index_parent_2].links[tid];
	links_child_2[tid] = population[index_parent_1].links[tid];
	}
	}

	if (tid + blockSize < size){
	if (tid + blockSize < link_crossover_index){
	links_child_1[tid + blockSize] = population[index_parent_1].links[tid + blockSize];
	links_child_2[tid + blockSize] = population[index_parent_2].links[tid + blockSize];
	}
	else{
	links_child_1[tid + blockSize] = population[index_parent_2].links[tid + blockSize];
	links_child_2[tid + blockSize] = population[index_parent_1].links[tid + blockSize];
	}
	}
	*/
	/*if (tid < rules){
	if (tid < rule_crossover_index){
	//	rules_child_1[tid] = population[index_parent_1].rules[tid];
	//	rules_child_2[tid] = population[index_parent_2].rules[tid];
	}
	else{
	//	rules_child_1[tid] = population[index_parent_2].rules[tid];
	rules_child_2[tid] = population[index_parent_1].rules[tid];
	}
	}*/

	__syncthreads();


	__shared__ float child_connectivity_1, child_connectivity_2;

	/** Child 1 **/
	calculeConnectivity<blockSize>(links_child_1, NODES*MAX_INPUTS, &child_connectivity_1);

	if (fabsf(child_connectivity_1 - INIT_CONNECTIVITY) < MAX_CONNECTIVITY_DISTANCE){
		// Links
		if (tid > link_crossover_index){
			population[indices[0]].links[tid] = links_child_1[tid];
		}
		if (blockSize >= 512 && ((tid + 512) > link_crossover_index) && (tid + 512) < INDIVIDUAL_SIZE){
			population[indices[0]].links[tid + 512] = links_child_1[tid + 512];
		}

		// Rules
		if (tid < RULES_PER_NODE*NODES && tid >= rule_crossover_index){
			population[indices[0]].rules[tid] = population[indices[1]].rules[tid];
		}
	}

	/** Child 2 **/
	calculeConnectivity<blockSize>(links_child_2, NODES*MAX_INPUTS, &child_connectivity_2);

	if (fabsf(child_connectivity_2 - INIT_CONNECTIVITY) < MAX_CONNECTIVITY_DISTANCE){
		// Links
		if (tid > link_crossover_index){
			population[indices[1]].links[tid] = links_child_2[tid];
		}
		if (blockSize >= 512 && ((tid + 512) > link_crossover_index) && (tid + 512) < INDIVIDUAL_SIZE){
			population[indices[1]].links[tid + 512] = links_child_2[tid + 512];
		}

		// Rules
		if (tid < RULES_PER_NODE*NODES && tid >= rule_crossover_index){
			population[indices[1]].rules[tid] = rules_child_2[tid];
		}
	}

}

__global__ void setup_indices(int* indices_best, thrust::device_ptr<int> positions){

	int offset = POPULATION / POPULATIONS;
	int index = (threadIdx.x + blockDim.x*blockIdx.x);
	unsigned int parents_per_population = ELITE_MEMBERS / POPULATIONS;
	int population = index / parents_per_population;

	if (index < (ELITE_MEMBERS)){
		indices_best[index] = positions[population*offset + index%parents_per_population];
	}

}

void total_fitness(thrust::device_ptr<float> keys, thrust::device_ptr<float> total_fitness){

	inverse<float> unary_op;
	thrust::plus<float> binary_op;
	float init = 0;

	for (int i = 0; i < (POPULATIONS); i++){
		total_fitness[i] = thrust::transform_reduce(keys + i*POPULATION / POPULATIONS, keys + (i + 1)*POPULATION / POPULATIONS, unary_op, init, binary_op);
	}

}



__global__ void roulette_selection(float pop_fitness[], int indices[], thrust::device_ptr<float> total_fitness, hiprandState *globalState){

	const unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	const unsigned int population_index = index / (POPULATION / POPULATIONS);
	const unsigned int pop_offset = POPULATION / POPULATIONS;

	if (index < POPULATION){
		float random = generate(
			globalState) * total_fitness[population_index];
		int pick = pop_offset*population_index;
		double offset = 0;
		float individual_fitness = 1 / pop_fitness[pick];
		while (random > offset){
			offset += powf(individual_fitness, 3);
			pick++;
		}
		indices[index] = pick % (POPULATION / POPULATIONS);
	}

}

__global__ void sus_selection(float pop_fitness[], int indices[], thrust::device_ptr<float> total_fitness, hiprandState *globalState){

	const unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	const unsigned int population_index = index / (POPULATION / POPULATIONS);
	const unsigned int pop_offset = POPULATION / POPULATIONS;

	__shared__ float randoms[POPULATIONS];

	if (index < POPULATION && (index % POPULATIONS) == 0){
		randoms[population_index] = generate(globalState);
	}

	__syncthreads();

	// Tener un random por  poblacion
	if (index < POPULATION){
		double p = total_fitness[population_index] / (POPULATION / POPULATIONS);
		int pick = pop_offset*population_index;
		double start = randoms[population_index] * p;
		double sum = 1 / pop_fitness[pick];
		double pointer = start + (index % (POPULATION / POPULATIONS)) * p;

		if (sum >= pointer){
			indices[index] = pick;
		}
		else{
			for (++pick; pick < POPULATION; pick++){
				sum += 1 / pop_fitness[pick];
				if (sum >= pointer){
					indices[index] = pick;
					break;
				}
			}
		}

	}

}

template <unsigned int elements_per_population, unsigned int offset>
__global__ void migrate_v2(network population[], int indices_best[]){

	const int index = blockIdx.x;
	const unsigned int tid = threadIdx.x;
	const unsigned int pop_index = index / elements_per_population;
	const unsigned int offset_elements = (POPULATION / POPULATIONS) * 2 - elements_per_population;

	if (index < ELEMENTS_TO_MIGRATE){
		unsigned int id_best = pop_index*offset + index%elements_per_population;
		unsigned int id_worst = (id_best + offset_elements) % ELITE_MEMBERS;
		unsigned int index_best = indices_best[id_best];
		unsigned int index_worst = indices_best[id_worst];

		population[index_worst + ((pop_index + 1)*POPULATION / POPULATIONS) % POPULATION].links[tid] = population[(pop_index*POPULATION / POPULATIONS) + index_best].links[tid];

		switch (blockDim.x){
		case 64:
			population[index_worst + ((pop_index + 1)*POPULATION / POPULATIONS) % POPULATION].links[tid + 64] = population[(pop_index*POPULATION / POPULATIONS) + index_best].links[tid + 64];
		case 128:
			population[index_worst + ((pop_index + 1)*POPULATION / POPULATIONS) % POPULATION].links[tid + 128] = population[(pop_index*POPULATION / POPULATIONS) + index_best].links[tid + 128];
		case 256:
			population[index_worst + ((pop_index + 1)*POPULATION / POPULATIONS) % POPULATION].links[tid + 256] = population[(pop_index*POPULATION / POPULATIONS) + index_best].links[tid + 256];
		case 512:
			population[index_worst + ((pop_index + 1)*POPULATION / POPULATIONS) % POPULATION].links[tid + 512] = population[(pop_index*POPULATION / POPULATIONS) + index_best].links[tid + 512];
			break;
		}

		if (tid < NODES){
			//	population[index_worst + ((pop_index + 1)*POPULATION / POPULATIONS) % POPULATION].nodes[tid] = population[(pop_index*POPULATION / POPULATIONS) + index_best].nodes[tid];
			population[index_worst + ((pop_index + 1)*POPULATION / POPULATIONS) % POPULATION].rules[tid] = population[(pop_index*POPULATION / POPULATIONS) + index_best].rules[tid];
		}

	}
}

void compute_fitness(hipStream_t stream1, hipStream_t stream2, network * d_population, float * link_distances, float * d_x_links, float * link_fitness, hipblasHandle_t handle, float * node_distances, float * d_x_nodes, float * node_fitness, float * d_current_fitness, char goal_links[], char init_nodes[], char goal_nodes[]){
	link_distance_v2<8> << <(POPULATION*NODES*MAX_INPUTS / 8 + 32 * 4 - 1) / (32 * 4), 32 * 4 >> >(d_population, link_distances, POPULATION, goal_links);
	gpuErrchk(hipPeekAtLastError());

	node_distance <4> << <(POPULATION*NODES + 4 * 32 - 1) / (32 * 4), 32 * 4 >> >(d_population, node_distances, POPULATION, init_nodes, goal_nodes);
	gpuErrchk(hipPeekAtLastError());

	sum_link_distances(link_distances, d_x_links, link_fitness, NODES*MAX_INPUTS, POPULATION, handle);
	gpuErrchk(hipPeekAtLastError());


	sum_node_distances(node_distances, d_x_nodes, node_fitness, NODES, POPULATION, handle);
	gpuErrchk(hipPeekAtLastError());

	pop_fitness(link_fitness, node_fitness, d_current_fitness, handle);
	gpuErrchk(hipPeekAtLastError());
}

void elite_selection(network * d_population, float * link_distances, float * d_x_links, float * link_fitness, hipblasHandle_t handle,
	float * node_distances, float * d_x_nodes, float * node_fitness, float * d_current_fitness, thrust::device_ptr<float> device_ptr_fitness,
	int * d_indices, thrust::device_ptr<int> dev_indices, int * d_indices_best){

	sort_population(device_ptr_fitness, dev_indices);
	gpuErrchk(hipPeekAtLastError());

	//if (ELITE_MEMBERS != POPULATION){
	setup_indices << <((ELITE_MEMBERS + 1023) * 20) / 1024, ELITE_MEMBERS / 16 >> >(d_indices_best, dev_indices);
	gpuErrchk(hipPeekAtLastError());
	//}

}

void roulette_sel(network * d_population, float * link_distances, float * d_x_links, float * link_fitness, hipblasHandle_t handle,
	float * node_distances, float * d_x_nodes, float * node_fitness, float * d_current_fitness, thrust::device_ptr<float> device_ptr_fitness,
	int * d_indices, thrust::device_ptr<int> dev_indices, int * d_indices_best, hiprandState* globalState, thrust::device_ptr<float> total_pop_fitness){

	total_fitness(device_ptr_fitness, total_pop_fitness);

	sus_selection << <(POPULATION + 512 + 1) / 512, 512 >> >(d_current_fitness, d_indices_best, total_pop_fitness, globalState);
}

void migrate(unsigned int current_generation, unsigned int migration_freq, network * d_population, int * d_indices_best){
	if (POPULATIONS > 1 && (current_generation%migration_freq) == 0){
		migrate_v2<ELEMENTS_TO_MIGRATE / POPULATIONS, ELITE_MEMBERS / POPULATIONS> << <ELEMENTS_TO_MIGRATE, 256 >> >(d_population, d_indices_best);
		gpuErrchk(hipPeekAtLastError());
	}
}

network population[POPULATION];

int main(void) {

	for (int e = 0; e < EXECUTIONS; e++){

		float time;
		hipEvent_t start, stop;

		HANDLE_ERROR(hipEventCreate(&start));
		HANDLE_ERROR(hipEventCreate(&stop));
		HANDLE_ERROR(hipEventRecord(start));

		FILE *f;

		hipError_t cudastat;
		hipblasStatus_t stat;

		hipblasHandle_t handle;
		stat = hipblasCreate(&handle);

		float *d_current_fitness;		// Holds total population fitness for each generation

		float* link_fitness;		// Link fitness of each individual
		float* node_fitness;		// Node fitness of each individual

		network h_BEST_INDIVIDUAL;
		float h_BEST_INDIVIDUAL_FITNESS_HOST = 1.0;
		float aux = 1.0;

		float * d_x_links, *d_x_nodes;	// Array to hold the whether a link/node exists or not in the goal network
		int *d_indices_best, *d_indices_worst;	// indices for the best and worst individuals
		float *node_distances, *link_distances;	// Node and link distances PER NETWORK

		network *d_population;	// Network population

		hiprandState* devStates;

		int null_links = thrust::count(INIT_NETWORK_HOST.links, INIT_NETWORK_HOST.links + NODES*MAX_INPUTS, 0);
		gpuErrchk(hipPeekAtLastError());
		const float init_connectivity = (NODES * MAX_INPUTS - (float)null_links) / (NODES * MAX_INPUTS);

		HANDLE_ERROR(
			hipMemcpyToSymbol(HIP_SYMBOL(INIT_CONNECTIVITY), &init_connectivity, sizeof(float)));


		char buf[0x100];
		_snprintf(buf, sizeof(buf), "P-Sexec%d-%s_pob%dpops%d_MIGRs%d_gen%dfreq%d.csv", e, "ELITE", POPULATION, POPULATIONS, ELEMENTS_TO_MIGRATE / POPULATIONS, GENERATIONS, MIGRATION_FREQUENCY);

		f = fopen(buf, "w");
		if (f == NULL)
		{
			printf("Error opening file!\n");
			exit(1);
		}

		HANDLE_ERROR(
			hipMemcpyToSymbol(HIP_SYMBOL(LAMBDA_VALUES), &LAMBDA_HOST, sizeof(float)*POPULATIONS));

		int * d_indices;
		HANDLE_ERROR(
			hipMalloc(&d_indices, POPULATION*sizeof(int)));

		thrust::device_ptr<int> dev_indices = thrust::device_pointer_cast(d_indices);

		HANDLE_ERROR(
			hipMalloc(&d_current_fitness, POPULATION*sizeof(float)));

		char * d_goal_links;
		HANDLE_ERROR(
			hipMalloc(&d_goal_links, INDIVIDUAL_SIZE*sizeof(char)));
		HANDLE_ERROR(
			hipMemcpy(d_goal_links, &GOAL_NETWORK_HOST.links, sizeof(char)*INDIVIDUAL_SIZE, hipMemcpyHostToDevice));

		char * d_init_nodes;
		HANDLE_ERROR(
			hipMalloc(&d_init_nodes, INDIVIDUAL_SIZE*sizeof(char)));
		HANDLE_ERROR(
			hipMemcpy(d_init_nodes, &INIT_NETWORK_HOST.nodes, sizeof(char)*NODES, hipMemcpyHostToDevice));

		char * d_goal_nodes;
		HANDLE_ERROR(
			hipMalloc(&d_goal_nodes, NODES*sizeof(char)));
		HANDLE_ERROR(
			hipMemcpy(d_goal_nodes, &GOAL_NETWORK_HOST.nodes, sizeof(char)*NODES, hipMemcpyHostToDevice));


		float * total_fitness;
		HANDLE_ERROR(
			hipMalloc(&total_fitness, POPULATIONS*sizeof(float)));
		thrust::device_ptr<float> dev_total_fitness = thrust::device_pointer_cast(total_fitness);
		thrust::device_ptr<float> device_ptr_fitness = thrust::device_pointer_cast(d_current_fitness);
		thrust::device_ptr<float> min_ptr = thrust::min_element(device_ptr_fitness, device_ptr_fitness + POPULATION);

		float* x_links = new float[NODES*MAX_INPUTS];

		for (int i = 0; i < NODES*MAX_INPUTS; i++)
		{
			x_links[i] = 1;//(1-LAMBDA)/(NODES*MAX_INPUTS);
		}

		float* x_nodes = new float[NODES];

		for (int i = 0; i < NODES; i++)
		{
			x_nodes[i] = 1;//(LAMBDA)/(NODES);
		}


		HANDLE_ERROR(
			hipMalloc(&link_fitness, POPULATION*sizeof(float)));
		HANDLE_ERROR(
			hipMalloc(&node_fitness, POPULATION*sizeof(float)));
		HANDLE_ERROR(
			hipMalloc(&d_x_links, NODES*MAX_INPUTS*sizeof(float)));
		HANDLE_ERROR(
			hipMemcpy(d_x_links, x_links, NODES*MAX_INPUTS*sizeof(float), hipMemcpyHostToDevice));

		HANDLE_ERROR(
			hipMalloc(&d_x_nodes, NODES*sizeof(float)));
		HANDLE_ERROR(
			hipMemcpy(d_x_nodes, x_nodes, NODES*sizeof(float), hipMemcpyHostToDevice));


		HANDLE_ERROR(
			hipMalloc(&d_indices_best, ELITE_MEMBERS*sizeof(int)));
		HANDLE_ERROR(
			hipMalloc(&d_indices_worst, ELITE_MEMBERS*sizeof(int)));

		HANDLE_ERROR(
			hipMalloc(&node_distances, POPULATION * NODES * sizeof(float)));
		HANDLE_ERROR(
			hipMalloc(&link_distances, POPULATION * NODES * MAX_INPUTS * sizeof(float)));

		HANDLE_ERROR(
			hipMalloc(&devStates, POPULATION * 1024 * sizeof(hiprandState)));

		print_parameters();

		setup_kernel_V3 << < POPULATION * 2, 512 >> > (devStates);
		gpuErrchk(hipPeekAtLastError());

		HANDLE_ERROR(
			hipMalloc(&d_population, POPULATION*sizeof(network)))
			;

		generate_individual << <(POPULATION + 1023) / 512, 512 >> >(d_population, NODES, MAX_INPUTS, RULES_PER_NODE, devStates);
		gpuErrchk(hipPeekAtLastError());

		const int individuals_block_ns = INDIVIDUALS_PER_BLOCK;

		int *link_crossover_indices, *rule_crossover_indices;
		HANDLE_ERROR(
			hipMalloc(&link_crossover_indices, (ELITE_MEMBERS / 2)*sizeof(int)));
		HANDLE_ERROR(
			hipMalloc(&rule_crossover_indices, (ELITE_MEMBERS / 2)*sizeof(int)));

		float* h_fit = (float*)malloc(POPULATION*sizeof(float));

		hipStream_t stream0, stream1, stream2, stream3, stream4;
		hipStreamCreate(&stream0);
		hipStreamCreate(&stream1);
		hipStreamCreate(&stream2);
		hipStreamCreate(&stream3);
		hipStreamCreate(&stream4);


		sequence << <(POPULATION + 1023) / 1024, 1024 >> >(d_indices);
		gpuErrchk(hipPeekAtLastError());

		generateLinkCrossoverindices << < (ELITE_MEMBERS / 2 + 447) / 448, 448, 0, stream2 >> >(link_crossover_indices, devStates);
		gpuErrchk(hipPeekAtLastError());

		for (int i = 0; i < GENERATIONS && h_BEST_INDIVIDUAL_FITNESS_HOST != 0; i++){

			compute_fitness(stream3, stream4, d_population, link_distances, d_x_links, link_fitness, handle, node_distances, d_x_nodes, node_fitness, d_current_fitness, d_goal_links, d_init_nodes, d_goal_nodes);

			min_v2(d_population, device_ptr_fitness, min_ptr, i, f, &h_BEST_INDIVIDUAL_FITNESS_HOST, &h_BEST_INDIVIDUAL);
			gpuErrchk(hipPeekAtLastError());

			elite_selection(d_population, link_distances, d_x_links, link_fitness, handle, node_distances, d_x_nodes, node_fitness, d_current_fitness, device_ptr_fitness, d_indices, dev_indices, d_indices_best);
			//roulette_sel(d_population, link_distances, d_x_links, link_fitness, handle, node_distances, d_x_nodes, node_fitness, d_current_fitness, device_ptr_fitness, d_indices, dev_indices, d_indices_best, devStates, dev_total_fitness);

			gpuErrchk(hipPeekAtLastError());

			print_progress((float)i / (float)GENERATIONS, h_BEST_INDIVIDUAL_FITNESS_HOST);
			gpuErrchk(hipPeekAtLastError());

			//generateRuleCrossoverindices << <(ELITE_MEMBERS / 2 + 31) / 32, 32 >> >(rule_crossover_indices, devStates);
			//gpuErrchk(hipPeekAtLastError());

			crossover<512, POPULATION / POPULATIONS> << <ELITE_MEMBERS / 2, (NODES*MAX_INPUTS) / 2, 0, stream2 >> >(d_population, d_indices_best, link_crossover_indices);
			gpuErrchk(hipPeekAtLastError());

			mutation << <(POPULATION + 32 * 5 + 1) / (32 * 5), 32 * 5, 0, stream0 >> >(d_population, NODES, MAX_INPUTS, RULES_PER_NODE, LINK_MUTATION_PROB, RULE_MUTATION_PROB, devStates);
			gpuErrchk(hipPeekAtLastError());

			sequence << <(POPULATION + 1023) / 1024, 1024, 0, stream1 >> >(d_indices);
			gpuErrchk(hipPeekAtLastError());
			generateLinkCrossoverindices << < (ELITE_MEMBERS / 2 + 447) / 448, 448, 0, stream2 >> >(link_crossover_indices, devStates);
			gpuErrchk(hipPeekAtLastError());

			migrate(i, MIGRATION_FREQUENCY, d_population, d_indices_best);

			//	gpuErrchk( hipDeviceSynchronize() );
		}
		// Retrieve data from device

		HANDLE_ERROR(hipEventRecord(stop));
		HANDLE_ERROR(hipEventSynchronize(stop));
		HANDLE_ERROR(hipEventElapsedTime(&time, start, stop));
		fprintf(f, "\nTime to generate:  %3.1f ms \n", time);
		print_network_file(h_BEST_INDIVIDUAL, f);

		fclose(f);

		hipFree(d_population);
		hipFree(total_fitness);
		hipFree(d_indices);
		hipFree(d_current_fitness);
		hipFree(link_fitness);
		hipFree(node_fitness);
		hipFree(d_x_links);
		hipFree(d_x_nodes);
		hipFree(d_indices_best);
		hipFree(node_distances);
		hipFree(link_distances);
		hipFree(devStates);
		hipStreamDestroy(stream0);
		hipStreamDestroy(stream1);
		hipStreamDestroy(stream2);
		hipStreamDestroy(stream3);
		hipStreamDestroy(stream4);
		hipblasDestroy(handle);

	}
	hipDeviceReset();
	return 0;
}